#include "hip/hip_runtime.h"
#if defined(__APPLE__)
#if _GLIBCXX_ATOMIC_BUILTINS == 1
#undef _GLIBCXX_ATOMIC_BUILTINS
#endif // _GLIBCXX_ATOMIC_BUILTINS
#endif // __APPLE__

#include <iostream>
#include <stdio.h>
#include <cutil.h>
#include <vector>
#include <string>

/*
 * Copyright (c) 2009 Steve Worley < m a t h g e e k@(my last name).com >
 *
 * Permission to use, copy, modify, and distribute this software for any
 * purpose with or without fee is hereby granted, provided that the above
 * copyright notice and this permission notice appear in all copies.
 *
 * THE SOFTWARE IS PROVIDED "AS IS" AND THE AUTHOR DISCLAIMS ALL WARRANTIES
 * WITH REGARD TO THIS SOFTWARE INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY AND FITNESS. IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR
 * ANY SPECIAL, DIRECT, INDIRECT, OR CONSEQUENTIAL DAMAGES OR ANY DAMAGES
 * WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS, WHETHER IN AN
 * ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS ACTION, ARISING OUT OF
 * OR IN CONNECTION WITH THE USE OR PERFORMANCE OF THIS SOFTWARE.
 */


/* SHA 1  brute force searcher for EngineYard hash match challenge.
   Steve Worley, July 18, 2009

   See the challenge: 
    http://www.engineyard.com/blog/2009/programming-contest-win-iphone-3gs-2k-cloud-credit/

   Discussion online on the NVidia CUDA pages!
   http://forums.nvidia.com/index.php?showtopic=102349


   Version 0.15. Check the forum post above for updates!

   Change history:
   
   v 0.16
      Now deletes timer
      -blocksize command line option for static block size

   v 0.15
      Detect if we're on G200 or not. On G200, we can use 192 threads. On G80/G90 we
      use a less efficient 128 threads.  CUDA 2.2 is way too register hungry!
   
   v 0.14
      Forum reminder printout for actual users
      Thread setting put at top of code (set to 128 to run with CUDA 2.2 and G80/G90)
      Rearranged first and third SHA f computes to save a clock on each

   v 0.13
      gcc compiler warning cleanups
      free memory at end of string search (slow leak of device mem)
      use a parallel reduction to find block best. 1% speedup.

   v 0.12 
      Now does a pre-hashed constant block to allow strings >64 characters. 
        In fact this is a requirement. This allows 12 words to be chosen.
      Updated hash speed, v11 didn't change computation for smaller block sizes
      5% speed boost by making simpler case for final 16 rounds where popping w[] is easier
      THIS CODE IS NOW USABLE FOR THE CONTEST. It's likely not final, but it will work!
      
   v 0.11: 
     added missing std:: prefix to vector and string for GCC
     Threads only iterate over 64*64 characters, not 93*93, to make smaller block work size
     #pragma unroll on the inner loops
     Now actually sets the device for multi-GPU
     KNOWN ISSUE: Still needs to pick only 12 prefix rules 

   v 0.10: 
     initial version


   PLEASE contribute to the code if you have ideas, optimizations or bugfixes.

   Yes, this means that if you win the contest, you can keep the EngineYard prize!
   But please TELL US by posting to the forum!


   Compile on windows with a line similar to:
      nvcc -I "E:\CUDA\common\inc"  gpusha1.cu  e:\cuda\common\lib\cutil32.lib -o gpusha1search.exe


   run with a commandline like:
         gpusha1search.exe -device 0 -blocksize 400 6cac827bae250971a8b1fb6e2a96676f7a077b60 Cloud Ruby DHH one eight six active record controller data rspec mongrel MySQL postgresSQL tokyo MRI jruby rubinius memcached exception metaprogramming reflection


   You can check the SHA1 hash of both the challenge and resulting best phrase with any SHA1 
   computation. A nice cut-and-paste one online is:
         http://sha1-hash-online.waraxe.us/

*/


/* For best performance, use 192 threads. But CUDA 2.2 and 2.3 compile this with huge 
   register use, causing kernel launch failures on G80 and G90 hardware.
   This can be avoided by compiling with CUDA 2.0, or reducing the line below to 
   be 128 threads. */

int threads=192;



int staticblocksize=0; // use dynamic block sizes by default


#ifdef __DEVICE_EMULATION__
#define debugprint printf
#define EMUSYNC __syncthreads()
#else
__device__ void NOOPfunction(char *format) {}
__device__ void NOOPfunction(char *format, unsigned int onearg) {}
__device__ void NOOPfunction(char *format, unsigned int onearg, unsigned int twoargs) {}
__device__ void NOOPfunction(char *format, char *onearg) {}
#define EMUSYNC do {} while (0)
#define debugprint NOOPfunction
#endif


__shared__ unsigned int keyString[16];
__shared__ unsigned int goalHash[5];
__shared__ unsigned int initVector[5];



/* extra complexity since SHA uses big-endian computes but x86 and CUDA use little-endian.
   We need to flip the words. But this will also flip the per-thread destination indices! */

__shared__ int firstCharIndex; // first character each thread can permute
__shared__ int secondCharIndex; // second character each thread can permute


__host__ __device__ unsigned int swapends(unsigned int v) 
{
  return 
    ((255&(v>> 0))<<24)+
    ((255&(v>> 8))<<16)+
    ((255&(v>>16))<<8)+
    ((255&(v>>24))<<0);
}

// when a byte index into a word array is word flipped for endianness, what's our new index?
__device__ unsigned int swappedIndex(int v)
{
  int remainder=v&3;
  return (v&0xFFFFFFFC)+(3-remainder);
}

__device__ void initStringAndGoal(const unsigned int *deviceKeystring,
				  int stringLength,  
				  int blockIndex,
				  const unsigned int *deviceInitVector,
				  const unsigned int *deviceGoalHash)
{
  char *charkeyString=(char *)keyString; // just for convienience

  if (threadIdx.x<16)
    keyString[threadIdx.x]=deviceKeystring[threadIdx.x];

  if (threadIdx.x<5) {    
    initVector[threadIdx.x]=deviceInitVector[threadIdx.x];
    goalHash[threadIdx.x]=deviceGoalHash[threadIdx.x];
  }
  
  EMUSYNC;
  
  if (threadIdx.x==0) {
    
    charkeyString[stringLength]=' '; // one space before our 5 characters
    
    // There are 93 printable ASCII chars (skipping space).
    // index is really three digits of a radix 93 number.
    // we initialize the first three chars based on this three digit number. 
    int c=blockIndex/(93*93);
    charkeyString[stringLength+1]= (char)(33+c); // 33 is first ASCII char, "!"

    blockIndex-=c*93*93;
    c=blockIndex/93;
    charkeyString[stringLength+2]= (char)(33+c);
    blockIndex-=c*93;
    c=blockIndex;
    charkeyString[stringLength+3]= (char)(33+c);

    /* chars 4, 5 will be set by individual threads in their own copies */
    
    charkeyString[stringLength+6]=(char)128; // SHA1 sets first non-data bit to '1'
    keyString[15]=512+8*(stringLength+6); // Length of string, +1 for space, +5 for appended chars

    // after big-endian flipping, which index do our two changable characters end up at?
    firstCharIndex=swappedIndex(stringLength+4);
    secondCharIndex=swappedIndex(stringLength+5);       
  }
  
  __syncthreads();
  
  //  if (threadIdx.x==0) 
  //    for (int i=0; i<16; i++) debugprint("M%ld %08x\n", i, keyString[i]);
  
  if (threadIdx.x<15)
    keyString[threadIdx.x]=swapends(keyString[threadIdx.x]);
 
  __syncthreads();
  
  //  for (int i=0; i<16; i++) debugprint("S%ld %08x\n", i, keyString[i]);
}


__device__ void prepareString(int trialIndex, int stringLength)
{
  extern __shared__ unsigned int fullw[];
  unsigned int *w=fullw+17*threadIdx.x; // spaced by 17 to avoid bank conflicts

  for (int i=0; i<16; ++i) w[i]=keyString[i];
  
  int c=trialIndex>>6; // 0 to 64
  ((char *)w)[firstCharIndex]=(char)(33+c);
  c=trialIndex&63;
  ((char *)w)[secondCharIndex]= (char)(33+c);

  //  if (threadIdx.x==0) debugprint("testing string -%s-\n", (char *)w);
}


/* We don't want to precompute and store all 80 w array
   values. Instead we store only the next 16 values and update them in
   a logrolling array. Complicated but it means we can fit the tables
   in shared memory */
__device__ unsigned int popNextW(unsigned int *w, int &wIndex)
{
  unsigned int nextW=w[wIndex&15];
  int thisIndex=wIndex&15;
  w[thisIndex]^=w[(wIndex+16-3)&15]^w[(wIndex+16-8)&15]^w[(wIndex+16-14)&15];
  w[thisIndex]=  (w[thisIndex]<<1) | (w[thisIndex]>>31);
  ++wIndex;

  //  if (threadIdx.x==0) debugprint("pop %08x\n", nextW);
  return nextW;
}

/* same as above but we don't need to compute more of the table  at the end. */
__device__ unsigned int popFinalWs(unsigned int *w, int &wIndex)
{
  unsigned int nextW=w[wIndex&15];
  ++wIndex;
  return nextW;
}



__device__ int computeSHAscore()
{
  extern __shared__ unsigned int fullw[];
  unsigned int *w=fullw+17*threadIdx.x; // spaced by 17 to avoid bank conflicts
  int wIndex=0;


  //  if (threadIdx.x==0) debugprint("-%s-\n", (char *)w);

  /* SHA algorithm. See
   http://en.wikipedia.org/wiki/SHA_hash_functions.  Big
   implementation difference is we use a rolling 16-entry table for the w[]
   array to save precious shared memory space */

  unsigned int a = initVector[0];
  unsigned int b = initVector[1];
  unsigned int c = initVector[2];
  unsigned int d = initVector[3];
  unsigned int e = initVector[4];

#pragma unroll 999
  for (int i=0; i<20; ++i) {
    unsigned int thisW=popNextW(w, wIndex);
    //    unsigned int f= (b&c)|((~b)&d);
    unsigned int f= d ^ (b & (c^d)); // alternate computation of above
    unsigned int temp=((a<<5)|(a>>27))+f+e+0x5A827999+thisW;
    e=d;
    d=c;
    c=(b<<30)|(b>>2);
    b=a;
    a=temp;
  }

#pragma unroll 999
  for (int i=20; i<40; ++i) {
    unsigned int thisW=popNextW(w, wIndex);
    unsigned int f= b^c^d;
    unsigned int temp=((a<<5)|(a>>27))+f+e+0x6ED9EBA1+thisW;
    e=d;
    d=c;
    c=(b<<30)|(b>>2);
    b=a;
    a=temp;
  }

#pragma unroll 999
  for (int i=40; i<60; ++i) {
    unsigned int thisW=popNextW(w, wIndex);
    //    unsigned int f= (b&c) | (b&d) | (c&d);
    unsigned int f= (b&c) | (d & (b|c)); // alternate computation of above
    unsigned int temp=((a<<5)|(a>>27))+f+e+0x8F1BBCDC+thisW;
    e=d;
    d=c;
    c=(b<<30)|(b>>2);
    b=a;
    a=temp;
  }

#pragma unroll 999
  for (int i=60; i<64; ++i) {
    unsigned int thisW=popNextW(w, wIndex);
    unsigned int f= b^c^d;
    unsigned int temp=((a<<5)|(a>>27))+f+e+0xCA62C1D6+thisW;
    e=d;
    d=c;
    c=(b<<30)|(b>>2);
    b=a;
    a=temp;
  }


#pragma unroll 999
  for (int i=64; i<80; ++i) {
    unsigned int thisW=popFinalWs(w, wIndex); // simpler compute for final rounds
    unsigned int f= b^c^d;
    unsigned int temp=((a<<5)|(a>>27))+f+e+0xCA62C1D6+thisW;
    e=d;
    d=c;
    c=(b<<30)|(b>>2);
    b=a;
    a=temp;
  }

  a+= initVector[0];
  b+= initVector[1];
  c+= initVector[2];
  d+= initVector[3];
  e+= initVector[4];

  /* the SHA hash is now a b c d e (concatinated) */

  //  if (threadIdx.x==0) debugprint("Hash= %08x %08x, %08x", a, b, c);
 

  /* xor with goal hash. Score is just summed population count. */

  int score=__popc(a^goalHash[0]) + 
    __popc(b^goalHash[1])+ +
    __popc(c^goalHash[2]) + 
    __popc(d^goalHash[3]) + 
    __popc(e^goalHash[4]); 

  return score;
}

__device__ void testSHA(int trialIndex, int stringLength, int &bestScore, int &bestIndex)
{
  prepareString(trialIndex, stringLength);
  int score=computeSHAscore();
  if (score<bestScore) {
    bestScore=score;
    bestIndex=trialIndex;
  }
}

__device__ void reportResultOfBestThread(int index, int score, unsigned int *bestarray)
{
  extern __shared__ unsigned int w[];

  /* each thread concatinates its best score with its best index */  
  w[threadIdx.x]= (score<<24)+index;
  __syncthreads();
  

  
  // I'm assuming blockdim.x< 256 here
  if (threadIdx.x+128< blockDim.x) w[threadIdx.x]=min(w[threadIdx.x], w[threadIdx.x+128]);
  __syncthreads();  
  
  if (threadIdx.x<64 && threadIdx.x+64< blockDim.x) w[threadIdx.x]=min(w[threadIdx.x], w[threadIdx.x+64]);
  __syncthreads();  

  if (threadIdx.x<32) { // no need for syncthreads in a single warp
    w[threadIdx.x]=min(w[threadIdx.x], w[threadIdx.x+32]); 
    w[threadIdx.x]=min(w[threadIdx.x], w[threadIdx.x+16]); 
    w[threadIdx.x]=min(w[threadIdx.x], w[threadIdx.x+8]); 
    w[threadIdx.x]=min(w[threadIdx.x], w[threadIdx.x+4]); 
    w[threadIdx.x]=min(w[threadIdx.x], w[threadIdx.x+2]); 
    //    w[threadIdx.x]=min(w[threadIdx.x], w[threadIdx.x+1]); 
  }
  
  if (threadIdx.x==0) bestarray[blockIdx.x]=min(w[0], w[1]);

}



/* Given a fixed string like "Rubinius one eight" with length LESS
   THAN 50 CHARACTERS, an 8-word goal hash value, and an absolute
   block number from 0 to 93*93*93-1=804356.  Evaluate all the possible appended 5
   character substrings (with first three characters enumerated over the
   given block range given by the starting number and the kernel's
   block count, last 2 characters enumerated over all 93^2
   possibilities).  Return value is an array of each blocks's local best result,
   which likely should be searched on the CPU for the minimum.

   The input keystring is loaded as 8 words. Pad your string at the end with 0s to populate it to
   16 words.
 */   

__global__ void sha1search(const unsigned int *deviceKeystring, 
			   int stringLength, 
			   const unsigned int *deviceInitVector, 
			   const unsigned int *deviceGoalHash, 
			   int blockIndexOffset, /* 0..93*93*93-1 */
			   unsigned int *bestarray)
{ 
  if (blockIndexOffset+blockIdx.x>=93*93*93) { // end of work
    bestarray[blockIdx.x]=0xFFFFFFFF;
    return;
  }

  initStringAndGoal(deviceKeystring, stringLength, blockIndexOffset+blockIdx.x, 
		    deviceInitVector, deviceGoalHash);

  int perThreadBestScore=9999;
  int perThreadBestIndex=0;
  int trialindex=threadIdx.x;

  while (trialindex<64*64) {   // iterate over the 4K  test hash strings
    testSHA(trialindex, stringLength, perThreadBestScore, perThreadBestIndex);    
    trialindex+=blockDim.x; 
  }
  
  __syncthreads(); // let all threads finish their looped work
  
  reportResultOfBestThread(perThreadBestIndex, perThreadBestScore, bestarray);
}


unsigned int initPopNextW(unsigned int *w, int &wIndex) 
{
  return w[wIndex++];
}


void initHash(const char *baseString, unsigned int h_InitVector[5])
{
  unsigned int w[80]={0};
  int wIndex=0;
  strncpy((char *)w, baseString, 512/8); 

  for (int i=0; i<16; i++) w[i]=swapends(w[i]);

  for (int i=16; i<80; i++) {
    w[i]=w[i-3]^w[i-8]^w[i-14]^w[i-16];
    w[i]=(w[i]<<1)|(w[i]>>31);
  }
	

  unsigned int a = 0x67452301;
  unsigned int b = 0xEFCDAB89;
  unsigned int c = 0x98BADCFE;
  unsigned int d = 0x10325476;
  unsigned int e = 0xC3D2E1F0;

  for (int i=0; i<20; ++i) {
    unsigned int thisW=initPopNextW(w, wIndex);
    unsigned int f= (b&c)|((~b)&d);
    unsigned int temp=((a<<5)|(a>>27))+f+e+0x5A827999+thisW;
    e=d;
    d=c;
    c=(b<<30)|(b>>2);
    b=a;
    a=temp;
  }

  for (int i=20; i<40; ++i) {
    unsigned int thisW=initPopNextW(w, wIndex);
    unsigned int f= b^c^d;
    unsigned int temp=((a<<5)|(a>>27))+f+e+0x6ED9EBA1+thisW;
    e=d;
    d=c;
    c=(b<<30)|(b>>2);
    b=a;
    a=temp;
  }

  for (int i=40; i<60; ++i) {
    unsigned int thisW=initPopNextW(w, wIndex);
    unsigned int f= (b&c) | (b&d) | (c&d);
    unsigned int temp=((a<<5)|(a>>27))+f+e+0x8F1BBCDC+thisW;
    e=d;
    d=c;
    c=(b<<30)|(b>>2);
    b=a;
    a=temp;
  }

  
  for (int i=60; i<80; ++i) {
    unsigned int thisW=initPopNextW(w, wIndex);
    unsigned int f= b^c^d;
    unsigned int temp=((a<<5)|(a>>27))+f+e+0xCA62C1D6+thisW;
    e=d;
    d=c;
    c=(b<<30)|(b>>2);
    b=a;
    a=temp;
  }

  a+= 0x67452301;
  b+= 0xEFCDAB89;
  c+= 0x98BADCFE;
  d+= 0x10325476;
  e+= 0xC3D2E1F0;

  h_InitVector[0] = a;
  h_InitVector[1] = b;
  h_InitVector[2] = c;
  h_InitVector[3] = d;
  h_InitVector[4] = e;

}


void searchSHAWithBaseString(const char *baseString,
			     const unsigned int h_GoalHash[5],
			     int &bestScore,
			     char *bestString)
{
  unsigned int h_Keystring[16]={0};
  unsigned int h_InitVector[5]={0};
  unsigned int *d_Keystring;
  const int maxBlocks=5000;
  unsigned int  h_Best[maxBlocks];
  unsigned int  *d_GoalHash;
  unsigned int  *d_InitVector;
  unsigned int  *d_Best;
  int blocksPerKernelCall=10; // we'll dynamically ramp this up to aim for 40ms kernels
  int startblock=0;
  int lastPrintTime=0;
  int forumReminder=0;
  unsigned int timer = 0;
  int stringlength=strlen(baseString)-64; // we don't count the first block's!
  CUT_SAFE_CALL( cutCreateTimer( &timer));

  sprintf((char *)h_Keystring, baseString+64); // skip over first (constant) chunk

  CUDA_SAFE_CALL( hipMalloc((void**)&d_Keystring, 16*32 ));
  CUDA_SAFE_CALL( hipMalloc((void**)&d_GoalHash, 5*32 ));
  CUDA_SAFE_CALL( hipMalloc((void**)&d_InitVector, 5*32 ));
  CUDA_SAFE_CALL( hipMalloc((void**)&d_Best, maxBlocks*32 ));
	
  initHash(baseString, h_InitVector); // do SHA1 hash of first (constant) block
  
  CUDA_SAFE_CALL( hipMemcpy( d_Keystring, h_Keystring, 16*32, hipMemcpyHostToDevice) );
  CUDA_SAFE_CALL( hipMemcpy( d_GoalHash, h_GoalHash, 5*32, hipMemcpyHostToDevice) );
  CUDA_SAFE_CALL( hipMemcpy( d_InitVector, h_InitVector, 5*32, hipMemcpyHostToDevice) );


  while (startblock<93*93*93) {
    CUT_SAFE_CALL( cutResetTimer( timer));
    CUT_SAFE_CALL( cutStartTimer( timer));

    if (staticblocksize>0) blocksPerKernelCall=staticblocksize;

    if (startblock+blocksPerKernelCall>93*93*93)
      blocksPerKernelCall=93*93*93-startblock; // don't go past end

    sha1search<<<blocksPerKernelCall, threads, threads*17*4>>>(d_Keystring, stringlength,
							       d_InitVector, 
							       d_GoalHash, 
							       startblock, 
							       d_Best);    
    CUDA_SAFE_CALL( hipMemcpy( &h_Best, d_Best, blocksPerKernelCall*sizeof(unsigned int), 
				hipMemcpyDeviceToHost) );
    CUT_SAFE_CALL( cutStopTimer( timer));
    
    float duration=cutGetTimerValue( timer);
   
    int bestblock=0;
    for (int i=1; i<blocksPerKernelCall; i++) if (h_Best[i]<h_Best[0]) {
	h_Best[0]=h_Best[i];
	bestblock=i;
      }
    
    int bestindex=h_Best[0]&0x00FFFFFF;
    bestblock+=startblock;
    
    if (h_Best[0]>>24 < bestScore) { // global best!
      bestScore=h_Best[0]>>24;
      sprintf(bestString, "%s %c%c%c%c%c",
	      baseString,
	      33+((bestblock/(93*93))%93),
	      33+((bestblock/93)%93),
	      33+((bestblock)%93),
	      33+((bestindex>>6)),
	      33+(bestindex&63));
    }
    
    int thistime=time(NULL);
    if (thistime>lastPrintTime+5) {
      lastPrintTime=thistime;

    printf( "Processing time: %d blocks in %0.2f ms. %0.3f megahashes/sec\n", 
	    blocksPerKernelCall, duration, 64*64*0.001*blocksPerKernelCall/duration);
 
      printf("  Best score for this pass is %d, block %d, index %d\n", 
	     h_Best[0]>>24, 
	     bestblock,
	     bestindex);
      
      printf("  %s %c%c%c%c%c\n",
	     baseString,
	     33+((bestblock/(93*93))%93),
	     33+((bestblock/93)%93),
	     33+((bestblock)%93),
	     33+((bestindex>>6)),
	     33+(bestindex&63));
            
      printf("Best score: %d  with\n%s\n", bestScore, bestString);

      if (forumReminder==0)
	printf("\n\nThis code is provided for free.\nIf you win the SHA1 contest, congratulations, the prize is YOURS!\nYou don't need to share! But please also visit the CUDA forum\nwhere we discuss this tool, and report your progress and/or success.\nVisit  http://forums.nvidia.com/index.php?showtopic=102349 . Thanks!\nCrashing, invalid 0 score, or wild behavior? Update your NVIDIA card drivers to the latest stable release version.\n");
      forumReminder++;
      if (forumReminder>4) forumReminder=0;
    }
    
    startblock+=blocksPerKernelCall;
    const float goalDuration=30.0; // 30 milliseconds 
    if (duration<0.1*goalDuration)      blocksPerKernelCall=2*blocksPerKernelCall+1;
    else if (duration<1.0*goalDuration) blocksPerKernelCall++;
    else if (duration<1.5*goalDuration) blocksPerKernelCall-=3;
    else if (duration<3.0*goalDuration) blocksPerKernelCall/=2;
    else blocksPerKernelCall=1+0.01*blocksPerKernelCall;
    
    if (blocksPerKernelCall>maxBlocks) blocksPerKernelCall=maxBlocks;
    if (blocksPerKernelCall<4) blocksPerKernelCall=4;
  }

  CUDA_SAFE_CALL(hipFree(d_Keystring));
  CUDA_SAFE_CALL(hipFree(d_GoalHash));
  CUDA_SAFE_CALL(hipFree(d_InitVector));
  CUDA_SAFE_CALL(hipFree(d_Best));
  CUT_SAFE_CALL(cutDeleteTimer(timer));

}

void usage()
{
    std::cout << "gpusha1search -device 1 -blocksize 200 (40 hex digits of goal hash) word1 word2 word3 ..." << std::endl;
    std::cout << "example: gpusha1search -device 0 AB23456789AB23456789AB23456789AB23456789 apple banana carrot" << std::endl;
    exit(1);
}


int fromhex(char c)
{
  if (c>='0' && c<='9') return c-'0';
  if (c>='A' && c<='F') return c-'A'+10;
  if (c>='a' && c<='f') return c-'a'+10;
  return -1;  
}


/* I'm sure there must be a more clever way of doing this */
void parseHash(char *hash, unsigned int GoalHash[5])
{
  char *nextchar=hash;
  unsigned char *g=(unsigned char *)GoalHash;
  for (int i=0; i<20; i++) { // pull off two digits at a time
    int a=fromhex(*(nextchar++));
    int b=fromhex(*(nextchar++));
    if (a<0 || b<0) {
      printf("%s doesn't look like a 40 digit hex hash.\n", hash);
      exit(0);
    }
    *(g++)=16*a+b;
  }
  for (int i=0; i<5; i++) GoalHash[i]=swapends(GoalHash[i]);
}

void parseCmdLine(int argc, char **argv, unsigned int GoalHash[5], std::vector<std::string> &dict)
{
  int deviceCount;                                                        
  CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));                
  if (deviceCount == 0) {                                                  
    fprintf(stderr, "Error: no devices supporting CUDA found.\n");		
    exit(1);							
  }									

  int device=0;

  if (std::string("-device")==argv[1]) {
    device=atol(argv[2]);
    if (device<0) device=0;
    if (device>deviceCount-1) device=deviceCount-1;
    argv+=2;
    argc-=2;
  }


  if (std::string("-blocksize")==argv[1]) {
    staticblocksize=atol(argv[2]);
    argv+=2;
    argc-=2;
  }


  if (argc<3) {
    usage();
  }
  
  if (argc<14) {
    printf("You must enter at least 12 dictionary words.\n");
    usage();
  } 

  hipDeviceProp_t deviceProp;                                               
  CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, device));
  fprintf(stderr, "Using device %d: %s\n", device, deviceProp.name);

  if (deviceProp.major < 1) {                                              
    fprintf(stderr, "Error: device does not support CUDA.\n");     
    exit(EXIT_FAILURE);                                                  
  }                                                                        
  
  hipSetDevice(device);
  
  // G200 has enough registers to run a full set of threads.
  // G80/G90 is register poor. CUDA 2.2 nvcc is too generous with register use,
  // but it does make faster code than CUDA 2.0
  if (deviceProp.regsPerBlock<10000) threads=128; else threads=192;

  parseHash(argv[1], GoalHash);
  
  for (int i=0; i<argc-2; ++i) 
    dict.push_back(argv[i+2]);

}

void randomPermute(unsigned int &seed, std::vector<std::string> &v)
{
  /* we COULD use the C++ random_shuffle here, but that's difficult to seed!
     It uses its own algorithm, and we want to add some time-based shuffle.
     So do a dumb set of random swaps. Not every permutation is equally likely,
     and the PRNG is not good, but it's more than enough for this kind of selection. */

  
  for (int i=0; i<v.size()-1; i++) {
    seed= 1103515245*seed+12345; // simple LCG
    unsigned int hashedseed=seed^(seed>>5)^(seed>>19)^(seed>>28);
    hashedseed*=0xDEADBEEF;
    hashedseed^=(hashedseed>>18);
    int swapindex=i+(hashedseed%(v.size()-i));
    std::swap(v[i], v[swapindex]);
  }
}

/* make a random selection of 12 keywords. */
/* NOTE FOR 0.11 this is WRONG for the contest, we need to make a two-block
   version to allow longer strings. This version is still valid but for speed testing.
*/

std::string GetRandomBaseString(unsigned int &seed, std::vector<std::string> &dict)
{
  std::string out;

  do {
    seed+=0xABCDEF;
    seed+=time(NULL);  
    
    randomPermute(seed, dict);   
    out=dict[0];
    for (int i=1; i<12; i++) out=out+" " +dict[i];
  } 
  while (out.length()<64 && out.length()>110);

  /* we're allowed to permute capitalization */
  for (int i=0; i<out.length(); i++) {
    seed= 1103515245*seed+12345; // simple LCG
    if (seed>>31) out[i]=toupper(out[i]);
    else out[i]=tolower(out[i]);
  }

  return out;
}

int main(int argc, char **argv) 
{
  char bestString[128]={0};
  int bestScore=999;
  unsigned int GoalHash[5]={0};
  unsigned int seed=time(NULL);
  std::vector<std::string> dict;
  
  if (argc<3) {
    usage();
  }

  parseCmdLine(argc, argv, GoalHash, dict);

  printf("\nSearching for hash %08x %08x %08x %08x %08x\n\nDictionary list:\n",
	 GoalHash[0], GoalHash[1], GoalHash[2], GoalHash[3], GoalHash[4]);

  for (int i=0; i<dict.size(); i++) std::cout << i << " " << dict[i] << std::endl;
  

  for (;;) {
    std::string base=GetRandomBaseString(seed, dict);
    std::cout << "Starting new pass with base string:" << base << std::endl;
    searchSHAWithBaseString(base.c_str(),
			    GoalHash,
			    bestScore, 
			    bestString);
  }
  
}
